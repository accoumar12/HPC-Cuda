
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// On notera BS directement dans le code plutôt qu'une valeur en dure
#define BS 16

void printMatrix(float *matrix, int size) {
	float *hostMatrix = (float *)malloc(size * size * sizeof(float));
	hipMemcpy(hostMatrix, matrix, size * size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%0.1f ", hostMatrix[i * size + j]);
		}
		printf("\n");
	}
	free(hostMatrix);
}

/** Retourne la différence (en secondes) entre deux timespec */
double get_delta(struct timespec begin, struct timespec end) {
	return end.tv_sec - begin.tv_sec + (end.tv_nsec - begin.tv_nsec) * 1e-9;
}

__global__ void mat_init(float *A, float *B, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * n + j;
    
    if (i < n && j < n) {
        if (i == j) {
            A[idx] = i;
        }
        B[idx] = j;
    }
}

__global__ void mat_mat(float * A, float * B, float * C, int n) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = i * n + j;
	
	if (i < n && j < n) {
		float sum = 0.0;
		for (int k=0; k<n; k++) {
			sum += A[n*i + k] * B[n*k+ j];
		}
		C[idx] = sum;
	}
}

__global__ void matmat_s(float *A, float *B, float *C, int n) {
	// Déclaration de la mémoire partagée pour les blocs de A et B
	__shared__ float sharedA[BS][BS];
	__shared__ float sharedB[BS][BS];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = i * n + j;

        float sum = 0.0;

	// Nombre de blocs nécessaires pour couvrir la matrice A et B
	int numBlocks = (n + BS - 1) / BS;

	for (int block = 0; block < numBlocks; ++block) {
	// Chargement du bloc de A et B en mémoire partagée
	if (i < n && block * BS + threadIdx.x < n) {
		sharedA[threadIdx.y][threadIdx.x] = A[i * n + block * BS + threadIdx.x];
	} 

	if (j < n && block * BS + threadIdx.y < n) {
		sharedB[threadIdx.y][threadIdx.x] = B[(block * BS + threadIdx.y) * n + j];
	}
	__syncthreads();

	// Calcul du produit scalaire du bloc en mémoire partagée
	for (int k = 0; k < BS; ++k) {
		sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
	}
	}

	// Écriture du résultat dans la matrice C
	if (i < n && j < n) {
		C[idx] = sum;
	}
}

int main(int argc, char * argv[]) {
	if (argc < 2) {
		printf("USAGE: %s <n>\n", argv[0]);
	}
	int n = atoi(argv[1]);
	int nb_blocks = (n + BS - 1) / BS;
	dim3 block_size(BS, BS);
	dim3 grid_size(nb_blocks, nb_blocks);
	struct timespec begin_matmat, end_matmat;

	// Allocations mémoires ...
	float * g_A, * g_B, * g_C;
	hipMalloc((void **)&g_A, n * n * sizeof(float));
	hipMalloc((void **)&g_B, n * n * sizeof(float));
	hipMalloc((void **)&g_C, n * n * sizeof(float));

	int nb_iter = 1;
	for (int i = 0; i < nb_iter; i++) {
		// Appels aux kernels mat_init et mat_mat
		mat_init<<<grid_size, block_size>>>(g_A, g_B, n);
		hipDeviceSynchronize();
		
		/*
		printMatrix(g_A, n);
		printf("\n");
		printMatrix(g_B, n);
		printf("\n");
		*/
		
		clock_gettime(CLOCK_REALTIME, &begin_matmat);
		mat_mat<<<grid_size, block_size>>>(g_A, g_B, g_C, n);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &end_matmat);
		printf("Temps du calcul matriciel sans mémoire partagée: %lf s.\n\n", get_delta(begin_matmat, end_matmat)); 
		
		clock_gettime(CLOCK_REALTIME, &begin_matmat);
		matmat_s<<<grid_size, block_size>>>(g_A, g_B, g_C, n);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &end_matmat);
		printf("Temps du calcul matriciel avec mémoire partagée: %lf s.\n\n", get_delta(begin_matmat, end_matmat));

		
		
		//printMatrix(g_C, n);
		//printf("\n");
	}
	hipFree(g_A);
	hipFree(g_B);
	

	float * h_C = (float*) malloc(n*n*sizeof(float));
	// Copie de C -> h_C
	hipMemcpy(h_C, g_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
	for (int i=0; i < n; i++) {
		for (int j=0; j < n; j++) {
			float expected = i * j;
			if (h_C[i*n+j] != expected) {
				printf("h_C[%d][%d] = %f != %f\n", i, j, h_C[i*n+j], expected);
			}
		}
	}

	// Libération de la mémoire
	free(h_C);
	hipFree(g_C);
}
